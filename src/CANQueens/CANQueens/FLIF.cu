#include "hip/hip_runtime.h"
/* Fatigue Leaky Integrate and Fire Neuron Class Source File
 * Parent class for Explore Memory and CA
 *
 * 200616
 * author = @ugurc
 * ugurcan.cakal@gmail.com
 */

#include "FLIF.cuh"


int FLIF::nextID = 0;

 // Inits

void FLIF::initFlags(int n, float activity, std::vector<bool>& flag_vec) {
    /* Initialize firing flags randomly
     *
     * Parameters:
     *      n(int):
     *          number of neurons
     *      activity(float):
     *          activity rate of neurons. 1.0 result in always fire.
     *      inhibitory(float):
     *          inhibitory neuron rate inside network.
     *          1.0 full inhibitory and 0.0 means full excitatory.
     *      flag_vec(std::vector<bool>&):
     *          reference to flag vector to be filled.
     */
    std::vector<bool>::iterator it;
    flag_vec.resize(n);
    
    for (it = flag_vec.begin(); it < flag_vec.end(); it++) {
        *it = (0 == (rand() % static_cast<int>(floorf(1.0f / activity))));
    }
}


void FLIF::initEF(int n, float upper, float lower, std::vector<float>& EF_vec) {
    /* Initialize energy/fatigueness levels randomly
     *
     * Parameters:
     *      n(int):
     *          number of neurons
     *      upper(float):
     *          upper bound for the level
     *      lower(float):
     *          lower bound for the level
     *      EF_vec(std::vector<float>&):
     *          reference to energy/fatigueness vector to be filled.
     */
    float temp;
    std::vector<float>::iterator it;
    EF_vec.resize(n);

    for (it = EF_vec.begin(); it < EF_vec.end(); it++) {
        temp = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
        temp *= (upper - lower);
        (*it) = temp - lower;
    }
}

// Updates
void FLIF::updateFlags(std::vector<bool>& flag_vec,
    const float& activity) {

    std::vector<bool>::iterator it;
    for (it = flag_vec.begin(); it < flag_vec.end(); it++) {
        *it = 0 == (rand() % static_cast<int>(floorf(1.0f / activity)));
    }
}


// Methods

std::string FLIF::dateTimeStamp(const char* filename) {
    /* Create an @ugurc format timestamp
     * For example the date 09 May 1995 and time 02:48:05
     * is encoded like year-month-day-hour-minute-second
     * 950509024805
     *
     * Parameters:
     *      filename (const char*):
     *          filename to be concatenated with the timestamp
     *
     * Returns:
     *		nameSTAMP(std::string):
     *			name with a dateTimeStamp like ugurc950509024805
     */

    time_t t = time(0);   // get time now
    struct tm* now = localtime(&t);
    char buffer[80];
    strftime(buffer, 80, "%y%m%d%H%M%S", now);
    std::string dateTime = buffer;
    return filename + dateTime;
}

int FLIF::num_fire(std::vector<bool>& firings) {
    /* Number of neurons fired in a given flag vector.
     *
     * Parameters:
     *      firings(std::vector<bool>&):
     *          firing flag vector consisting of booleans
     *
     * Returns:
     *      num(int):
     *          total number of fire
     */
    int fire = 0;
    std::vector<bool>::iterator it;
    for (it = firings.begin(); it < firings.end(); it++) {
        if (*it) {
            fire++;
        }
    }
    return fire;
}

REC_SIZE FLIF::sizeCheckRecord(int stop, int start) {
    REC_SIZE temp;
    temp.check = true;
    //std::cout << "SIZE CHECK" << std::endl;
    //std::cout << record[0].energy.size() << std::endl;
    int range = stop - start;
    if (start < 0) {
        std::cout << "Starting point cannot be less than 0!" << std::endl;
        start = 0;
        range = stop - start;
        std::cout << "Starting point has fixed to " << start << std::endl;
    }
    if (stop < start) {
        std::cout << "End point cannot be less than start " << start << "!" << std::endl;
        stop = start + abs(range);
        range = stop - start;
        std::cout << "End point has fixed to " << stop << std::endl;
    }

    if (stop > record.size()) {
        std::cout << "Not enough record to print " << stop << "(st/nd/rd/th) step!" << std::endl;
        stop = record.size();
        std::cout << "End point has fixed to " << stop << std::endl;
        if (stop - range > 0) {
            start = stop - range;
        }
        else {
            start = 0;
        }
        range = stop - start;
        std::cout << "Starting point has fixed to " << start << std::endl;
    }

    if (range <= 0) {
        std::cout << "Range is " << range << "!" << std::endl;
        temp.check = false;
    }

    temp.start = start;
    temp.stop = stop;
    return temp;
}

REC FLIF::setRecord(int available) {
    REC temp;
    temp.available = available;
    if ((available | 0b0111) == 0b1111) {
        //std::cout << (this->flags).size() << std::endl;
        temp.flags = this->flags;
    }

    if ((available | 0b1011) == 0b1111) {
        //std::cout << "rec2" << std::endl;
        temp.energy = this->energy;
    }

    if ((available | 0b1101) == 0b1111) {
        //std::cout << "rec3" << std::endl;
        temp.fatigue = this->fatigue;
    }

    if ((available | 0b1110) == 0b1111) {
        //std::cout << "rec4" << std::endl;
        temp.weights = this->weights;
    }
    
    return temp;
}

template<typename T>
std::string FLIF::vectorToString(const std::vector<T>& vec) {
    /* Convert a vector of U type to an std::string using
     * " " as delimiter.
     * 
     * Parameters:
     *      vec(std::vector<U>&):
     *          vector to be printed
     *
     * Returns:
     *      vec_string(std::string):
     *          string form of the vector
     */
    std::string temp = "";
    typename std::vector<T>::const_iterator it;
    for (it = vec.begin(); it < vec.end(); it++) {
        temp += std::to_string(*it) + " ";
    }
    return temp;
}

template<typename T>
void FLIF::vectorToCSV(std::ostream& file, const std::vector<T>& entry) {
    typename std::vector<T>::const_iterator it;
    bool start = true;
    for (it = entry.begin(); it < entry.end(); it++) {
        if (start) {
            file << *it;
            start = false;
        }
        else {
            file << " , " << *it;
        }
    }
    file << std::endl;
}

// Constructors

FLIF::FLIF() {
    ID = ++nextID;
    n_neuron = 0;
    activity = 0.0f;
    connectivity = 0.0f;
    inhibitory = 0.0f;
}

FLIF::~FLIF() {
}

std::string FLIF::getRecord(int timeStep) {
    std::string temp = "\n";

    temp += "timeStep " + std::to_string(timeStep) + "\n";

    if ((record[timeStep].available | 0b0111) == 0b1111) {
        temp += "\nFlags ["+ std::to_string(activity) +"] : ";
        temp += "(" + std::to_string(num_fire(record[timeStep].flags)) +
            "/" + std::to_string(n_neuron) + ")\n";
        temp += vectorToString<bool>(record[timeStep].flags);
    }
    
    if ((record[timeStep].available | 0b1011) == 0b1111) {
        temp += "\n\nEnergy Levels \n";
        temp += vectorToString<float>(record[timeStep].energy);
    }
    
    if ((record[timeStep].available | 0b1101) == 0b1111) {
        temp += "\n\nFatigue Levels \n";
        temp += vectorToString<float>(record[timeStep].fatigue);
    }

    if ((record[timeStep].available | 0b1110) == 0b1111) {
        temp += "\n\nWeights \n";
        std::vector<std::vector<float>>::iterator it_w;
        for (it_w = record[timeStep].weights.begin(); it_w < record[timeStep].weights.end(); it_w++) {
            temp += "|" + vectorToString<float>(*it_w) + "|\n";
        }
    }
    temp += "\n";
    return temp;
}

std::string FLIF::getActivity(int stop, int start) {

    if (stop == -1) {
        stop = record.size();
    }

    REC_SIZE rec = sizeCheckRecord(stop, start);
    start = rec.start;
    stop = rec.stop;
    // Size Check
    if (!rec.check) {
        std::cout << "Activity cannot be shown!" << std::endl;
        return "NA";
    }

    std::string temp = "\n";
    temp += "CA ID: " + std::to_string(getID()) + "\n";

    for (int i = start; i < stop; i++) {
        temp+=  getRecord(i) + "\n";
    }

    return temp;

}

std::string FLIF::getRaster(float threshold, int stop, int start) {
    /* Construct the string representing whole raster plot
     * for given time interval.
     *
     *  N_ID    ||         SPIKE ACTIVITY
     *  --------------------------------------------
     *  0       ||      |
     *  1       ||      |               |
     *  2       ||      |                       |
     *  3       ||
     *  --------------------------------------------
     *  TIME    ||      0       1       2       3
     *  --------------------------------------------
     *  FIRE    ||      2       0       1       1
     *  --------------------------------------------
     *  IGNIT   ||      1       0       1       1
     *
     * Parameters:
     *      start(int):
     *          starting timestep
     *      stop(int):
     *          ending timestep
     *      threshold(float):
     *          minimum rate of firing to ignit (0 by default)
     *          to show the ignit line, it must be greater than 0.
     *
     * Returns:
     *      raster(std::string):
     *          raster plot
     */
    if (stop == -1) {
        stop = record.size();
    }

    int range = stop - start;
    std::string temp = "\n";
    int n_threshold = threshold * n_neuron;
    
    REC_SIZE rec = sizeCheckRecord(stop, start);
    start = rec.start;
    stop = rec.stop;

    // Size Check
    if (!rec.check) {
        std::cout << "Raster cannot be plotted!" << std::endl;
        return "NA";
    }

    if ((record[0].available | 0b0111) != 0b1111) {
        std::cout << "No firing record available!" << std::endl
            << "Raster cannot be plotted!" << std::endl;
        return "NA";
    }

    // Header
    temp += "  \t";
    temp += " \n";
    temp += "N_ID \t||";
    temp += std::string(3 * (range), ' ');
    temp += "SPIKE ACTIVITY\n";
    temp += std::string(8 * (range + 1) + 4, '-') + "\n";

    // Body
    for (int i = 0; i < n_neuron; i++) {
        temp += std::to_string(i);
        temp += "\t||\t";
        for (int j = start; j < stop; j++) {
            if (record[j].flags[i]) {
                temp += "|\t";
            }
            else {
                temp += " \t";
            }
        }
        temp += "\n";
    }

    temp += std::string(8 * (range + 1) + 4, '-') + "\n";
    temp += "TIME \t||\t";
    for (int i = start; i < stop; i++) {
        temp += std::to_string(i) + "\t";
    }
    temp += "\n";
    temp += std::string(8 * (range + 1) + 4, '-') + "\n";
    temp += "FIRE \t||\t";
    for (int i = start; i < stop; i++) {
        temp += std::to_string(num_fire(record[i].flags)) + "\t";
    }
    temp += "\n";
    temp += std::string(8 * (range + 1) + 4, '-') + "\n";
    if (threshold > 0.0f) {
        temp += "IGNIT \t||\t";
        for (int i = start; i < stop; i++) {
            temp += std::to_string(num_fire(record[i].flags) >= n_threshold) + "\t";
        }
    }
    
    return temp;
}

//void FLIF::getRasterCSV(char* filename, float threshold, int stop, int start) {
//    /* Construct the string representing whole raster plot
//     * for given time interval.
//     *
//     *  N_ID    ||         SPIKE ACTIVITY
//     *  --------------------------------------------
//     *  0       ||      |
//     *  1       ||      |               |
//     *  2       ||      |                       |
//     *  3       ||
//     *  --------------------------------------------
//     *  TIME    ||      0       1       2       3
//     *  --------------------------------------------
//     *  FIRE    ||      2       0       1       1
//     *  --------------------------------------------
//     *  IGNIT   ||      1       0       1       1
//     *
//     * Parameters:
//     *      start(int):
//     *          starting timestep
//     *      stop(int):
//     *          ending timestep
//     *      threshold(float):
//     *          minimum rate of firing to ignit (0 by default)
//     *          to show the ignit line, it must be greater than 0.
//     *
//     * Returns:
//     *      raster(std::string):
//     *          raster plot
//     */
//    if (stop == -1) {
//        stop = record.size();
//    }
//
//    int range = stop - start;
//    int n_threshold = threshold * n_neuron;
//
//    REC_SIZE rec = sizeCheckRecord(stop, start);
//    start = rec.start;
//    stop = rec.stop;
//
//    // Size Check
//    if (!rec.check) {
//        std::cout << "Raster cannot be plotted!" << std::endl;
//        return ;
//    }
//
//    if ((record[0].available | 0b0111) != 0b1111) {
//        std::cout << "No firing record available!" << std::endl
//            << "Raster cannot be plotted!" << std::endl;
//        return ;
//    }
//
//    // File
//    std::string name = std::string(filename) + "_rasterID" +std::to_string(getID()) +".csv";
//    std::ofstream file(name, std::ofstream::out);
//
//    if (file.is_open()) {
//        // Header
//
//        file << "N_ID" << " , " << std::endl;
//
//        // Body
//        for (int i = 0; i < n_neuron; i++) {
//            file << i << " , ";
//            for (int j = start; j < stop; j++) {
//                if (record[j].flags[i]) {
//                    file << "x" << " , ";
//                }
//                else {
//                    file << " " << " , ";
//                }
//            }
//            file << " " << std::endl;
//        }
//        file << "TIME" << " , ";
//        for (int i = start; i < stop; i++) {
//            file << i << " , ";
//        }
//        file << std::endl;
//        file << "FIRE" << " , ";
//        for (int i = start; i < stop; i++) {
//            file << num_fire(record[i].flags) << " , ";
//        }
//        file << std::endl;
//        if (threshold > 0.0f) {
//            file << "IGNIT" << " , ";
//            for (int i = start; i < stop; i++) {
//                file << std::to_string(num_fire(record[i].flags) >= n_threshold) << " , ";
//            }
//        }
//    }
//    else {
//        std::cout << "Raster CSV file cannot open!" << std::endl;
//    }
//    file.close();
//
//}

void FLIF::getCSV(char* filename, int type, float threshold, int stop, int start) {

    if (stop == -1) {
        stop = record.size();
    }

    int range = stop - start;
    int n_threshold = threshold * n_neuron;
    REC_SIZE rec = sizeCheckRecord(stop, start);
    start = rec.start;
    stop = rec.stop;

    // Size Check
    if (!rec.check) {
        std::cout << "EF cannot be plotted!" << std::endl;
        return;
    }

    // File
    std::string specifier = "specifier";
    if (type == 0) {
        specifier = "_raster";
    }
    if (type == 1) {
        specifier = "_energy";
    }
    if (type == 2) {
        specifier = "_fatigue";
    }
    std::string name = std::string(filename) + specifier + ".csv";
    std::ofstream file(name, std::ofstream::out);

    if (file.is_open()) {
        // Header

        file << "N_ID" << " , " << std::endl;

        // Body
        for (int i = 0; i < n_neuron; i++) {
            file << i << " , ";
            for (int j = start; j < stop; j++) {
                if (type == 0) {
                    if (record[j].flags[i]) {
                        file << "x" << " , ";
                    }
                    else {
                        file << " " << " , ";
                    }
                }
                if (type == 1) {
                    file << record[j].energy[i] << " , ";
                }
                else if (type == 2) {
                    file << record[j].fatigue[i] << " , ";
                }
            }
            file << " " << std::endl;
        }
        file << "TIME" << " , ";
        for (int i = start; i < stop; i++) {
            file << i << " , ";
        }
        file << std::endl;
        if (type == 0) {
            file << "FIRE" << " , ";
            for (int i = start; i < stop; i++) {
                file << num_fire(record[i].flags) << " , ";
            }
            file << std::endl;
            if (threshold > 0.0f) {
                file << "IGNIT" << " , ";
                for (int i = start; i < stop; i++) {
                    file << std::to_string(num_fire(record[i].flags) >= n_threshold) << " , ";
                }
            }
        }
        
    }
    else {
        std::cout << "Type " << type << " file cannot open!" << std::endl;
    }
    file.close();

}

void FLIF::saveRecord(char* filename, float threshold, int stop, int start) {
    /* Save the record and the raster plot constructed by 
     * getRaster() and getRecord() recursively
     * to a .txt file, into ./test/ folder.
     *
     * Parameters:
     *      filename(char*):
     *          filename to be stamped then used
     *      start(int):
     *          starting timestep
     *      stop(int):
     *          ending timestep
     *      threshold(float):
     *          minimum rate of firing to ignit (0 by default)
     *          to show the ignit line, it must be greater than 0.
     */
    
    std::string raster_name = "./test/raster_" + dateTimeStamp(filename) + ".txt";
    std::ofstream raster_file(raster_name, std::ofstream::out);

    std::string record_name = "./test/record_" + dateTimeStamp(filename) + ".txt";
    std::ofstream record_file(record_name, std::ofstream::out);

    if (stop == -1) {
        stop = record.size();
    }

    REC_SIZE rec = sizeCheckRecord(stop, start);
    start = rec.start;
    stop = rec.stop;
    // Size Check
    if (!rec.check) {
        std::cout << "Record cannot be saved!" << std::endl;
        return;
    }

    if ((record[0].available | 0b0111) != 0b1111) {
        std::cout << "No firing record available!" << std::endl
            << "Raster cannot be plotted!" << std::endl;
    }
    else {
        if (raster_file.is_open()) {
            raster_file << getRaster(threshold, stop, start) << std::endl;
        }
        else {
            std::cout << "Raster file cannot open!" << std::endl;
        }
        raster_file.close();
    }

    if (record_file.is_open()) {
        for (int i = start; i < stop; i++) {
            record_file << getRecord(i) << std::endl;
        }
        record_file.close();
    }
    else {
        std::cout << "Record file cannot open!" << std::endl;
    }
}

void FLIF::getWeightCSV(char* filename, int stop, int start) {
    std::string name = std::string(filename) + "_weight.csv";
    std::vector <int> num;
    std::vector<int>::iterator it_num;
    int counter = 0;

    std::ofstream file(name, std::ofstream::out);
    std::vector<std::vector<float>>::iterator it;
    int outgoing;
    if (file.is_open()) {
        
        for (int i = start; i < stop; i++) {
            file << "t = " << i << std::endl;
            file << "N_ID / N_ID" << " , ";
            counter = 0;
            num.resize(record[i].weights[0].size());
            for (it_num = num.begin(); it_num < num.end(); it_num++) {
                *it_num = counter;
                counter++;
            }

            vectorToCSV<int>(file, num);
            outgoing = 0;
            for (it = record[i].weights.begin(); it < record[i].weights.end(); it++) {
                file << outgoing << " , ";
                vectorToCSV<float>(file, *it);
                outgoing++;
            }
            file << std::endl;
        }
    }
    else {
        std::cout << "Weights file cannot open!" << std::endl;
    }
    file.close();
}

void FLIF::saveCSV(char* filename, float threshold, int stop, int start) {

    if (stop == -1) {
        stop = record.size();
    }

    if ((record[0].available | 0b0111) == 0b1111) { // FLAGS 
        getCSV(filename, 0, threshold, stop, start);
    }

    if ((record[0].available | 0b1011) == 0b1111) { // ENERGY
        getCSV(filename, 1, threshold, stop, start);
    }

    if ((record[0].available | 0b1101) == 0b1111) { // FATIGUE
        getCSV(filename, 2, threshold, stop, start);
    }

    if ((record[0].available | 0b1110) == 0b1111) { // WEIGHT
        getWeightCSV(filename, stop, start);
    }
}

int FLIF::getID(){
    /* ID getter
     *
     * Returns:
     *      ID(int):
     *          ID of the network starting from 1
     */
    return ID;
}

int FLIF::getN(){
    /* n_neuron getter
     *
     * Returns:
     *      n_neuron(int):
     *          Number of neurons inside network
     */
    return n_neuron;
}

std::string FLIF::getInfo() {
    return "HI\n";
}

// Set
void FLIF::setActivity(float act) {
    this->activity = act;
}

