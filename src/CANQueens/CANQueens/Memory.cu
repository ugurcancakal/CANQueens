/* Memory Class Source File
 *
 * 200517
 * author = @ugurc
 * ugurcan.cakal@gmail.com
 */

#include "Memory.cuh"

int Memory::d_n_neuron = 10;
float Memory::d_activity = 0.5f;
float Memory::d_connectivity = 1.0f;
float Memory::d_inhibitory = 0.2f;
float Memory::d_alpha = 0.2f;
int Memory::d_available = 0b1001;

Memory::Memory(int n, float activity, float connectivity, float inhibitory, float alpha) {
    //std::cout << "Memory constructed" << std::endl;

    this->n_neuron = n;
    this->activity = activity;
    this->connectivity = connectivity;
    this->inhibitory = inhibitory;

    this -> alpha = alpha;
    this -> w_average = 1.0f;
    this -> w_current = 1.0f;

    // Neurons
    //initFlags(n, activity, this->flags);
    initFlags(n, activity, this->h_flags);
    initFlags(n, activity, this->h_preFlags);
    initFlags(n, activity, this->h_postFlags);
    this->incomingList.push_back(this);
    this->outgoingList.push_back(this);

    //initWeights(n, n, connectivity, inhibitory, this->weights);
    initWeights(n, n, connectivity, inhibitory, this->h_weights);
}

Memory::~Memory() {
    std::cout << "Memory destructed" << std::endl;
}


// Running
void Memory::runFor_CPU(int timeStep, int available) {
    /* Run the CA for defined timestep and record the activity
     * Implemented for raster plot drawing
     *
     * Parameters:
     *      timestep(int):
     *          number of steps to stop running
     */
    for (int i = 0; i < timeStep; i++) {
        CSCToDense(this->weights, this->h_weights);
        record.push_back(setRecord(available));
        update_CPU();
    }

}

// Running
void Memory::runFor_GPU(int timeStep, int available) {
    /* Run the CA for defined timestep and record the activity
     * Implemented for raster plot drawing
     *
     * Parameters:
     *      timestep(int):
     *          number of steps to stop running
     */
    for (int i = 0; i < timeStep; i++) {
        getDeviceToHostCSC(this->h_weights, this->d_weights);
        hipMemcpy(this->h_flags, this->d_flags, (this->n_neuron) * sizeof(bool), hipMemcpyDeviceToHost);
        CSCToDense(this->weights, this->h_weights);

        record.push_back(setRecord(available));
        update_GPU();
    }

}

void Memory::update_CPU(float act) {
    /* Update the CA by updating neuron related data structures
     * ! pre_synaptic and post_synaptic not in use
     */
    if (act >= 0.0f && act < 1.0f) {
        setActivity(act);
    }
    updatePre(this->h_preFlags, this->preSize, this->incomingList);
    updatePost(this->h_postFlags, this->postSize, this->outgoingList);
    //updateWeights(this->weights, this->flags, this->flags, this->alpha, this->w_average, this->w_current);
    updateWeights(this->h_weights, this->preSize, this->h_preFlags, this->postSize, this->h_postFlags, this->alpha, this->w_average, this->w_current);

    //updateFlags(this->flags, this->activity);
    updateFlags(this->n_neuron, this->h_flags, this->activity);

}


void Memory::update_GPU(float act) {
    /* Update the CA by updating neuron related data structures
     * ! pre_synaptic and post_synaptic not in use
     */

    hipError_t cudaStatus;
    updatePre(this->h_preFlags, this->preSize, this->incomingList);
    cudaStatus = freeBoolDevice(this->d_preFlags);
    cudaStatus = initBoolDevice(this->preSize, this->d_preFlags, this->h_preFlags);
    
    updatePost(this->h_postFlags, this->postSize, this->outgoingList);
    cudaStatus = freeBoolDevice(this->d_postFlags);
    cudaStatus = initBoolDevice(this->postSize, this->d_postFlags, this->h_postFlags);
    
    dim3 gridSize = this->postSize;
    dim3 blockSize = this->preSize; // Limitted to 1024.

    if (act >= 0.0f && act < 1.0f) {
        setActivity(act);
    }
    updateWeights_kernel << <gridSize, blockSize >> > (this->preSize,
        this->d_preFlags,
        this->postSize,
        this->d_postFlags,
        this->alpha,
        this->w_average,
        this->w_current,
        this->d_weights->CO,
        this->d_weights->RI,
        this->d_weights->data);

    gridSize = 1;
    blockSize = this->n_neuron;

    updateFlags_kernel << <gridSize, blockSize >> > (this->n_neuron, this->d_flags, this->activity);
}

void Memory::POC_CPU() {
    int timeStep = 10;
    std::cout << "MEMORY CPU" << std::endl;

    Memory* myMEM;
    myMEM = new Memory(10);

    myMEM->runFor_CPU(timeStep);
    myMEM->setActivity(0.1);
    myMEM->runFor_CPU(timeStep);

    std::cout << myMEM->getActivity() << std::endl;
}

void Memory::POC_GPU() {
    int timeStep = 10;
    std::cout << "MEMORY GPU" << std::endl;

    Memory* myMEM;
    myMEM = new Memory(10);

    myMEM->initMemoryGPU();

    myMEM->runFor_GPU(timeStep);
    myMEM->setActivity(0.1);

    myMEM->runFor_GPU(timeStep);
    std::cout << myMEM->getActivity() << std::endl;
}

void Memory::initMemoryGPU() {
    this->initBoolDevice(this->n_neuron, this->d_flags, this->h_flags);
    this->initBoolDevice(this->n_neuron, this->d_preFlags, this->h_preFlags);
    this->initBoolDevice(this->n_neuron, this->d_postFlags, this->h_postFlags);
    this->initCSCDevice(this->d_weights, this->h_weights);
}