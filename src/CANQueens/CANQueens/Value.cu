#include "hip/hip_runtime.h"
/* Value Class Source File
 *
 * 200517
 * author = @ugurc
 * ugurcan.cakal@gmail.com
 */

#include "Value.cuh"

Value::Value(int n) {
    row = n;
    //std::cout << "Value constructed" << std::endl;
}

Value::~Value() {
    std::cout << "Value destructed" << std::endl;
}

std::string Value::toString() {
    return "Value";
}

void Value::update(int* chromosome) {
    std::cout << fitness(chromosome) << std::endl;
}

float Value::activity(int n, int* chromosome) {
    return (fitness(chromosome)*1.0f) / (maxCollision(n)*1.0f);
}

int Value::maxCollision(int n) {
    return ((n * (n - 1)) / 2);
}

int Value::fitness(int* chromosome) {
    // KONTROL ET
    /*int collision = 0;
    int d;
    for (int i = row; i >= 0; i--) {
        for (int k = i - 1; k >= 0; k--) {
            d = abs(chromosome[i] - chromosome[k]);
            if ((d == 0) || (d == i - k)) {
                collision++;
            }
        }
    }
    return collision;*/
    int collision = 0;
    int d;
    for (int i = 0; i < row; i++) {
        //std::cout << "i: " << i << std::endl;
        if (chromosome[i] >= row) {
            collision += 666;
        }
        for (int k = i+1; k < row; k++) {
            //std::cout << "k: " << k << std::endl;
            d = abs(chromosome[k] - chromosome[i]);
            if ((d == 0) || (d == k-i)) {
                collision++;
            }
        }
    }
    return collision;
}
