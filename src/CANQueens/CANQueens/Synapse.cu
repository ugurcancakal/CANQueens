#include "hip/hip_runtime.h"
/* Synapse Class Source File
 *
 * 200619
 * author = @ugurc
 * ugurcan.cakal@gmail.com
 */

#include "Synapse.cuh"

__global__ void updateWeights_kernel(const int pre_size,
    bool* const d_preFlags,
    const int post_size,
    bool* const d_postFlags,
    const float alpha,
    const float w_average,
    const float w_current,
    int* const CO,
    int* const RI,
    float* data) {

    // NO NEW CONNECTIONS ALLOWED FOR NOW

    unsigned int index = threadIdx.x + blockDim.x * blockIdx.x;
    unsigned int stride = blockDim.x * gridDim.x;
    unsigned int pre_index;
    unsigned int post_index;
    float delta;
    float tempData;
    float sign;
    int c;

    while (index < pre_size * post_size) {
        sign = 1.0f;
        // Indexing
        //pre_index = index / post_size;
        //post_index = index - (pre_index * post_size);   

        post_index = index / pre_size;
        pre_index = index - (post_index * pre_size);

        if (d_preFlags[pre_index]) {
            delta = 0.0f;
            // GET CSC DATA 
            tempData = 0.0f;
            if (CO[post_index] == CO[post_index + 1]) {
                tempData = 0.0f;
            }
            else {
                for (c = CO[post_index]; c < CO[post_index + 1]; c++) {
                    if (RI[c] == pre_index) {
                        tempData = data[c];
                        break;
                    }
                }
            }
            if (tempData != 0.0f) {
                sign = tempData / abs(tempData);
                if (d_postFlags[post_index]) {
                    delta = alpha * (1.0f - abs(tempData)) * expf(w_average - w_current);
                }
                else {
                    delta = (-1.0f) * alpha * abs(tempData) * expf(w_current - w_average);
                }
                data[c] += sign * delta;
                //data[index] = index;
            }
        }
        index += stride;
    }
}

Synapse::Synapse() {
    //std::cout << "Synapse constructed" << std::endl;
    connectivity = 0.0f;
    inhibitory = 0.0f;
    alpha = 0.0f; // learning rate
    w_average = 0.0f; // constant representing average total synaptic strength of the pre-synaptic neuron.
    w_current = 0.0f; // current total synaptic strength
    //n_neuron = 6;
    //initWeights(n_neuron, n_neuron, 0.2f, 0.0f, this->h_weights);
}

Synapse::~Synapse() {
    //std::cout << "Synapse destructed" << std::endl;
}

void Synapse::initWeights(int in, int out, float connectivity, float inhibitory, std::vector<std::vector<float>>& weight_vec) {
    /* Initialize neuron weights randomly
     * Sign of the weigth is determined by the inhibitory neuron rate
     *
     * An example connection map: (10x10, 0.2 inhibitory, 1.0 connectivity)
     * -------------------
     * | - - + + + + + + | <-- incoming line
     * | - - + + + + + + |
     * | - - + + + + + + |
     * | - - + + + + + + |
     * | - - + + + + + + |
     * | - - + + + + + + |
     * | - - + + + + + + |
     * | - - + + + + + + |
     * -------------------
     * 0<w<1
     *
     * Parameters:
     *      in(int):
     *          incoming connections. in = 10 creates 10 rows
     *      out(int):
     *          outgoing connections. out = 10 creates 10 columns
     *      connectivity(float):
     *          connectivity ratio inside network.
     *          1.0 means fully connected.
     *      inhibitory(float):
     *          inhibitory neuron rate inside network.
     *          1.0 full inhibitory and 0.0 means full excitatory.
     *      weight_vec(std::vector<std::vector<float>>&):
     *          reference to weight vector to be filled.
     */
    int n_inh;
    if (inhibitory > 0) {
        n_inh = static_cast<int>(floorf(1.0f / inhibitory));
    }
    else {
        n_inh = -1;
    }
    float sign = -1.0f;
    weight_vec.resize(in);
    std::vector<std::vector<float>>::iterator it;
    for (it = weight_vec.begin(); it < weight_vec.end(); it++) {
        (*it).resize(out);
    }

    // Connectivity range check
    if (connectivity < 0.0f) {
        connectivity = 0.0f;
    }
    else if (connectivity > 1.0f) {
        connectivity = 1.0f;
    }

    // Iterators
    std::vector<std::vector<float>>::iterator it_w;
    std::vector<float>::iterator it_weight;

    for (it_w = weight_vec.begin(); it_w < weight_vec.end(); it_w++) {
        for (it_weight = (*it_w).begin(); it_weight < (*it_w).end(); it_weight++) {
            if (n_inh > 0) {
                sign = (rand() % n_inh) == 0 ? -1.0f : 1.0f;
            }
            else {
                sign = 1.0f;
            }
            if (static_cast <float> (rand()) / static_cast <float> (RAND_MAX) < connectivity) {
                *it_weight = sign * static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
            }
            else {
                *it_weight = 0.0f;
            }
        }
    }
}

void Synapse::initWeights(int in, int out, float connectivity, float inhibitory, CSC*& h_weights) {
    /* 
     * CO :
     * 0 1 1 2 3 4 4
     * RI :
     * 1 2 2 3
     * Data :
     * 0.634938 0.427015 0.15772 0.113651
     *
     * DENSE :
     * |0.000000 0.000000 0.000000 0.000000 0.000000 0.000000 |
     * |0.634938 0.000000 0.000000 0.000000 0.000000 0.000000 |
     * |0.000000 0.000000 0.427015 0.157720 0.000000 0.000000 |
     * |0.000000 0.000000 0.000000 0.000000 0.113651 0.000000 |
     * (2,3): 0.15772
     */

    int n_inh;
    float temp;
    float sign = -1.0f;
    
    std::vector<COO> tempWeights;
    //std::cout << "INIT WEIGHTS" << std::endl;

    if (inhibitory > 0) {
        n_inh = static_cast<int>(floorf(1.0f / inhibitory));
    }
    else {
        n_inh = -1;
    }
    // Connectivity range check
    if (connectivity < 0.0f) {
        connectivity = 0.0f;
    }
    else if (connectivity > 1.0f) {
        connectivity = 1.0f;
    }

    for (int j = 0; j < out; j++) {
        for (int i = 0; i < in; i++) {
            if (n_inh > 0) {
                sign = (rand() % n_inh) == 0 ? -1.0f : 1.0f;
            }
            else {
                sign = 1.0f;
            }
            if (static_cast <float> (rand()) / static_cast <float> (RAND_MAX) < connectivity) {
                temp = sign * static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
                tempWeights.push_back(COO{ i, j, temp });
            }
        }
    }
    COOToCSC(h_weights, tempWeights, in, out);
}

void Synapse::COOToCSC(CSC*& target, const std::vector<COO>& source,int row, int col) {
    //target = new CSC();
    //target->rowSize = row;
    //target->columnSize = col;
    //target->nonzeros = source.size();
    //target->CO = new int[col + 1];
    //target->RI = new int[source.size()];
    //target->data = new float[source.size()];
    target = initCSC(row, col, source.size());

    int k = 0;
    int prevInd = 0;
    int counter = 1;

    target->CO[0] = 0;

    for (k = 0; k < *(target->nonzeros); k++) {
        while (source[k].j != prevInd) {
            target->CO[counter] = k;
            prevInd++;
            counter++;
        }
        target->RI[k] = source[k].i; 
        target->data[k] = source[k].data;
    }
    if (prevInd < col) {
        while (prevInd != col) {
            target->CO[counter] = k;
            prevInd++;
            counter++;
        }
    }
}

void Synapse::CSCToDense(std::vector<std::vector<float>>& target, CSC*& const source) {
    int indice = 0;
    int counter = 0;

    target.resize(*(source->rowSize));
    std::vector<std::vector<float>>::iterator it;
    for (it = target.begin(); it < target.end(); it++) {
        (*it).resize(*(source->columnSize));
    }

    for (int j = 0; j < *(source->columnSize); j++) {
        for (int i = 0; i < *(source->rowSize); i++) {  
            target[i][j] = getDataCSC(source, i, j);
        }
    }
}

float Synapse::getDataCSC(CSC*& target, int i, int j)
{
    if (target->CO[j] == target->CO[j + 1]){
        return 0.0f;
    }
    int colOff = target->CO[j];
    int colMax = target->CO[j + 1];
    for (int c = colOff; c < colMax; c++) {
        if (target->RI[c] == i) {
            return target->data[c];
        }
    }
    return 0.0f;
}

void Synapse::setDataCSC(CSC*& target, int i, int j, const float& data) {
    if (target->CO[j] == target->CO[j + 1]) {
        std::cout << "Empty cell cannot be set by this method" << std::endl;
        return;
    }
    int colOff = target->CO[j];
    int colMax = target->CO[j + 1];
    for (int c = colOff; c < colMax; c++) {
        if (target->RI[c] == i) {
            target->data[c] = data;
            return;
        }
    }
    std::cout << "Empty cell cannot be set by this method" << std::endl;
}

CSC* Synapse::initCSC(int rowSize, int columnSize, int nonzeros)
{
    CSC* target = new CSC();
    target->rowSize = new int(rowSize);
    target->columnSize = new int(columnSize);
    target->nonzeros = new int(nonzeros);
    target->CO = new int[columnSize + 1];
    target->RI = new int[nonzeros];
    target->data = new float[nonzeros];
    return target;
}

void Synapse::deleteCSC(CSC*& target) {
    delete target->rowSize;
    delete target->columnSize;
    delete target->nonzeros;
    delete[] target->CO;
    delete[] target->RI;
    delete[] target->data;
    delete target;
}

hipError_t Synapse::initCSCDevice(CSC*& d_CSC, CSC*& const h_CSC, bool allocHost, bool alloc) {
    //hipMalloc((void**)&d_CSC, sizeof(CSC));
    hipError_t cudaStatus;
    if (allocHost) {
        d_CSC = new CSC(); // this will store 6 device pointers on host.
    }

    if (alloc) {
        cudaStatus = hipMalloc((void**)&(d_CSC->rowSize), sizeof(int));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "CSC rowSize hipMalloc failed!");
            return cudaStatus;
        }
    }
    cudaStatus = hipMemcpy((d_CSC->rowSize), (h_CSC->rowSize), sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "CSC rowsize memcopy h2d failed!");
        return cudaStatus;
    }
    
    

    if (alloc) {
        cudaStatus = hipMalloc((void**)&(d_CSC->columnSize), sizeof(int));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "CSC columnSize hipMalloc failed!");
            return cudaStatus;
        }
    }
    cudaStatus = hipMemcpy((d_CSC->columnSize), (h_CSC->columnSize), sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "CSC columnsize memcopy h2d failed!");
        return cudaStatus;
    }



    if (alloc) {
        cudaStatus = hipMalloc((void**)&(d_CSC->nonzeros), sizeof(int));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "CSC nonzeros hipMalloc failed!");
            return cudaStatus;
        }
    } 
    cudaStatus = hipMemcpy((d_CSC->nonzeros), (h_CSC->nonzeros), sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "CSC columnsize memcopy h2d failed!");
        return cudaStatus;
    }


    if (alloc) {
        cudaStatus = hipMalloc((void**)&(d_CSC->CO), (*(h_CSC->columnSize) + 1) * sizeof(int));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "CSC CO hipMalloc failed!");
            return cudaStatus;
        }
    }
    
    cudaStatus = hipMemcpy((d_CSC->CO), (h_CSC->CO), (*(h_CSC->columnSize) + 1) * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "CSC CO memcopy h2d failed!");
        return cudaStatus;
    }



    if (alloc) {
        cudaStatus = hipMalloc((void**)&(d_CSC->RI), (*(h_CSC->nonzeros)) * sizeof(int));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "CSC RI hipMalloc failed!");
            return cudaStatus;
        }
    }
    
    cudaStatus = hipMemcpy((d_CSC->RI), (h_CSC->RI), (*(h_CSC->nonzeros)) * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "CSC RI memcopy h2d failed!");
        return cudaStatus;
    }

    if (alloc) {
        cudaStatus = hipMalloc((void**)&(d_CSC->data), (*(h_CSC->nonzeros)) * sizeof(float));
        if (cudaStatus != hipSuccess) {
            fprintf(stderr, "CSC data hipMalloc failed!");
            return cudaStatus;
        }
    }
    
    cudaStatus = hipMemcpy((d_CSC->data), (h_CSC->data), (*(h_CSC->nonzeros)) * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "CSC data memcopy h2d failed!");
        return cudaStatus;
    }
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after initCSCDevice!\n", cudaStatus);
        return cudaStatus;
    }
    return cudaStatus;
}

hipError_t Synapse::freeCSCDevice(CSC*& d_CSC) {
    hipError_t cudaStatus;

    cudaStatus = hipFree(d_CSC->rowSize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "CSC rowSize hipFree failed!");
        return cudaStatus;
    }
    cudaStatus = hipFree(d_CSC->columnSize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "CSC rowSize hipFree failed!");
        return cudaStatus;
    }
    cudaStatus = hipFree(d_CSC->nonzeros);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "CSC rowSize hipFree failed!");
        return cudaStatus;
    }
    cudaStatus = hipFree(d_CSC->CO);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "CSC rowSize hipFree failed!");
        return cudaStatus;
    }
    cudaStatus = hipFree(d_CSC->RI);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "CSC rowSize hipFree failed!");
        return cudaStatus;
    }
    cudaStatus = hipFree(d_CSC->data);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "CSC rowSize hipFree failed!");
        return cudaStatus;
    }
    delete d_CSC;

    return cudaStatus;
}

hipError_t Synapse::getDeviceToHostCSC(CSC*& h_CSC, CSC*& const d_CSC) {
    hipError_t cudaStatus;
    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(h_CSC->rowSize, d_CSC->rowSize, sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "CSC rowSize hipMemcpy failed!");
        return cudaStatus;
    }

    cudaStatus = hipMemcpy(h_CSC->columnSize, d_CSC->columnSize, sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "CSC columnSize hipMemcpy failed!");
        return cudaStatus;
    }

    cudaStatus = hipMemcpy(h_CSC->nonzeros, d_CSC->nonzeros, sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "CSC nonzeros hipMemcpy failed!");
        return cudaStatus;
    }

    cudaStatus = hipMemcpy(h_CSC->CO, d_CSC->CO, (*(h_CSC->columnSize) + 1) * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "CSC CO hipMemcpy failed!");
        return cudaStatus;
    }

    cudaStatus = hipMemcpy(h_CSC->RI, d_CSC->RI, (*(h_CSC->nonzeros)) * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "CSC RI hipMemcpy failed!");
        return cudaStatus;
    }

    cudaStatus = hipMemcpy(h_CSC->data, d_CSC->data, (*(h_CSC->nonzeros)) * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "CSC data hipMemcpy failed!");
        return cudaStatus;
    }

    return cudaStatus;
}

void Synapse::updateWeights(std::vector<std::vector<float>>& weight_vec,
    const std::vector<bool>& pre_vec,
    const std::vector<bool>& post_vec,
    const float& alpha,
    const float& w_average,
    const float& w_current) {
    /* Update weights of the FLIF neurons inside CA
     * according to hebbian learning rule.
     * That is, neurons fire together, wire together.
     * !! W-current W-average updates are to be done
     * !! w must be between 0<w<1
     * !! Changes incoming weights
     * !! it need to trace all incoming flags
     * !! it_f need to trace all outgoing flags rather than just internal
     * !! the weight between neurons fire together will increase in absolute value
     */
    float delta = 0.0f;
    float sign = 1.0f;
    // Size check
    if (weight_vec[0].size() != pre_vec.size()) {
        std::cout << "Weight matrix width is different than pre synaptic vector size!" << std::endl;
        //return;
    }
    if (weight_vec.size() != post_vec.size()) {
        std::cout << "Weight matrix height is different than post synaptic vector size!" << std::endl;
        //return;
    }

    // Iterators
    std::vector<bool>::const_iterator it_pre;
    std::vector<bool>::const_iterator it_post;
    std::vector<float>::iterator it_weight;
    std::vector<std::vector<float>>::iterator it_w;

    it_post = post_vec.begin();

    for (it_w = weight_vec.begin(); it_w < weight_vec.end(); it_w++) {
        it_pre = pre_vec.begin();
        for (it_weight = (*it_w).begin(); it_weight < (*it_w).end(); it_weight++) {
            if (*it_pre) {
                if ((*it_weight) != 0.0f) {
                    sign = (*it_weight) / abs(*it_weight);
                    if (*it_post) {
                        delta = alpha * (1.0f - abs(*it_weight)) * exp(w_average - w_current);
                    }
                    else {
                        delta = (-1.0f) * alpha * abs(*it_weight) * exp(w_current - w_average);
                    }
                    *it_weight += sign * delta;
                } 
            }
            it_pre++;
        }
        it_post++;
    }
}

void Synapse::updateWeights(CSC*& h_weights,
                            const int& preSize,
                            bool*& const h_preFlags,
                            const int& postSize,
                            bool*& const h_postFlags,
                            const float& alpha,
                            const float& w_average,
                            const float& w_current) {
    std::vector<COO> tempWeights;
    float delta = 0.0f;
    float sign = 1.0f;
    float temp;
    float tempData;

    if (*(h_weights->columnSize) != postSize) {
        std::cout << "Weight matrix width(" << *(h_weights->columnSize) 
            <<") is different than post synaptic vector size"<<(postSize)
            <<"!" << std::endl;
        //return;
    }
    if (*(h_weights->rowSize) != preSize) {
        std::cout << "Weight matrix height(" << *(h_weights->rowSize) 
            <<") is different than pre synaptic vector size(" << preSize 
            <<")!" << std::endl;
        //return;
    }
    //for (int j = 0; j < preSize; j++) {
    //    for (int i = 0; i < postSize; i++) {
    //        tempData = getDataCSC(h_weights, i, j);
    //        if (h_preFlags[j]) {
    //            if (tempData > 0.0f) {
    //                sign = tempData / abs(tempData);
    //            }
    //            if (h_postFlags[i]) {  
    //                delta = alpha * (1.0f - abs(tempData)) * exp(w_average - w_current);
    //            }
    //            else {
    //                delta = (-1.0f) * alpha * abs(tempData) * exp(w_current - w_average);
    //            }
    //            temp = tempData + (sign * delta);
    //            if (temp != 0.0f) {
    //                tempWeights.push_back(COO{ i, j, temp });
    //            }      
    //        }
    //        else {
    //            if (tempData != 0.0f) {
    //                tempWeights.push_back(COO{ i, j, tempData });
    //            }
    //        }
    //    }
    //    
    //}
    //
    //int in = *(h_weights->rowSize);
    //int out = *(h_weights->columnSize);
    //deleteCSC(h_weights);
    //COOToCSC(h_weights, tempWeights, in, out);

    for (int i = 0; i < preSize; i++) {
        for (int j = 0; j < postSize; j++) {
            tempData = getDataCSC(h_weights, i, j);
            if (h_preFlags[i]) {
                if (tempData != 0.0f) {
                    sign = tempData / abs(tempData);
                    if (h_postFlags[i]) {
                        delta = alpha * (1.0f - abs(tempData)) * exp(w_average - w_current);
                    }
                    else {
                        delta = (-1.0f) * alpha * abs(tempData) * exp(w_current - w_average);
                    }
                    setDataCSC(h_weights, i, j, tempData + (sign * delta));
                }
            }
        }
    }
}

void Synapse::updatePre(std::vector<bool>& pre_synaptic_flags,
    const std::vector<FLIF*>& incoming)
{
    std::vector<FLIF*>::const_iterator it;
    pre_synaptic_flags.clear();

    //std::cout << "\nINCOMING SIZE: " << incoming.size() << std::endl;
    for (it = incoming.begin(); it < incoming.end(); it++) {
        //std::cout << "PRE SIZE: " << pre_synaptic_flags.size() << std::endl;
        pre_synaptic_flags.insert(pre_synaptic_flags.end(),
            (*it)->flags.begin(),
            (*it)->flags.end());
    }

    //std::cout << "Updated SIZE: " << pre_synaptic_flags.size() << std::endl;
    //std::cout << getID() << " PRE:\n" << vectorToString<bool>(pre_synaptic_flags) << std::endl;
}

void Synapse::updatePost(std::vector<bool>& post_synaptic_flags,
    const std::vector<FLIF*>& outgoing) {

    std::vector<FLIF*>::const_iterator it;
    post_synaptic_flags.clear();

    //std::cout << "\nOUTGOING SIZE: " << outgoing.size() << std::endl;
    for (it = outgoing.begin(); it < outgoing.end(); it++) {
        //std::cout << "POST SIZE: " << post_synaptic_flags.size() << std::endl;
        post_synaptic_flags.insert(post_synaptic_flags.end(),
            (*it)->flags.begin(),
            (*it)->flags.end());
    }

    //std::cout << "Updated SIZE: " << post_synaptic_flags.size() << std::endl;
    //std::cout << getID() << " PRE:\n" << vectorToString<bool>(post_synaptic_flags) << std::endl;
}

void Synapse::updatePre(bool*& h_preFlags, int& preSize, const std::vector<FLIF*>& incoming) {
    std::vector<FLIF*>::const_iterator it;
    std::vector<bool>::iterator it_f;
    std::vector<bool> pre_synaptic_flags;
    int i = 0;
    delete[] h_preFlags;
    //std::cout << "\nINCOMING SIZE: " << incoming.size() << std::endl;
    for (it = incoming.begin(); it < incoming.end(); it++) {
        //std::cout << "PRE SIZE: " << pre_synaptic_flags.size() << std::endl;
        pre_synaptic_flags.insert(pre_synaptic_flags.end(),
            (*it)->h_flags,
            (*it)->h_flags + (*it)->n_neuron);
    }

    preSize = pre_synaptic_flags.size();
    //std::cout << "PRE SIZE: " << preSize << std::endl;
    h_preFlags = new bool[preSize];

    for (it_f = pre_synaptic_flags.begin(); it_f < pre_synaptic_flags.end(); it_f++) {
        h_preFlags[i] = *it_f;
        i++;
    }

}

void Synapse::updatePost(bool*& const h_postFlags, int& postSize, const std::vector<FLIF*>& outgoing) {
    std::vector<FLIF*>::const_iterator it;
    std::vector<bool>::iterator it_f;
    std::vector<bool> post_synaptic_flags;
    int i = 0;
    delete[] h_postFlags;
    //std::cout << "\nOUTGOING SIZE: " << outgoing.size() << std::endl;
    for (it = outgoing.begin(); it < outgoing.end(); it++) {
        //std::cout << "POST SIZE: " << post_synaptic_flags.size() << std::endl;
        post_synaptic_flags.insert(post_synaptic_flags.end(),
            (*it)->h_flags,
            (*it)->h_flags + (*it)->n_neuron);
    }

    postSize = post_synaptic_flags.size();
    
    h_postFlags = new bool[postSize];

    for (it_f = post_synaptic_flags.begin(); it_f < post_synaptic_flags.end(); it_f++) {
        h_postFlags[i] = *it_f;
        i++;
    }
}

// Connect
void Synapse::addIncomingWeights(std::vector<std::vector<float>>& resting,
    const std::vector<std::vector<float>>& in) {

    std::vector<std::vector<float>>::iterator it_w = resting.begin();
    std::vector<std::vector<float>>::const_iterator it_in = in.begin();
    std::vector<float> zeros((*it_in).size(), 0.0);

    if (resting.size() >= in.size()) {
        for (it_in = in.begin(); it_in < in.end(); it_in++) {
            (*it_w).insert((*it_w).end(), (*it_in).begin(), (*it_in).end());
            it_w++;
        }

        for (it_w = it_w; it_w < resting.end(); it_w++) {
            (*it_w).insert((*it_w).end(), zeros.begin(), zeros.end());
        }
    }
    else {
        std::cout << "Smaller Outgoing Vector!" << std::endl;
        for (it_w = resting.begin(); it_w < resting.end(); it_w++) {
            (*it_w).insert((*it_w).end(), (*it_in).begin(), (*it_in).end());
            it_in++;
        }
    }
}

void Synapse::addOutgoingWeights(std::vector<std::vector<float>>& resting,
    const std::vector<std::vector<float>>& out) {

    std::vector<std::vector<float>>::const_iterator it_out;
    std::vector<float> zeros(resting[resting.size() - 1].size() - out[0].size(), 0.0);
    std::vector<float> temp;

    if (resting[resting.size() - 1].size() >= out[0].size()) {
        for (it_out = out.begin(); it_out < out.end(); it_out++) {
            temp = *it_out;
            temp.insert(temp.end(), zeros.begin(), zeros.end());
            resting.push_back(temp);
        }
    }
    else {
        std::cout << "Smaller Incoming Vector!" << std::endl;
        int maxSize = resting[resting.size() - 1].size();
        for (it_out = out.begin(); it_out < out.end(); it_out++) {
            std::vector<float> newVec((*it_out).begin(), (*it_out).begin() + maxSize);
            resting.push_back(newVec);
        }
    }
}


void Synapse::addIncomingWeights(CSC*& resting, CSC*& const in) {
    // Be sure that incoming column size is equal or smaller
    int row_OFF = *(resting->rowSize);
    int row = *(resting->rowSize) + *(in->rowSize);
    int col = *(resting->columnSize);
    int nonzeros = *(resting->nonzeros) + *(in->nonzeros);

    int counter = 0;

    CSC* target = initCSC(row, col, nonzeros);

    int j = 0;
    int k = 0;
    for (int i = 0; i < col + 1; i++) {
        target->CO[i] = counter;
        for (j = resting->CO[i]; j < resting->CO[i + 1]; j++) {
            target->RI[j + k] = resting->RI[j];
            target->data[j + k] = resting->data[j];
            counter++;
        }
        if (i < *(in->columnSize)) { // check if in is smaller
            for (k = in->CO[i]; k < in->CO[i + 1]; k++) {
                target->RI[j + k] = row_OFF + in->RI[k];
                target->data[j + k] = in->data[k];
                counter++;
            }
        }
    }

    deleteCSC(resting);
    resting = target;
    
}

void Synapse::addOutgoingWeights(CSC*& resting, CSC*& const out) {
    // Be sure that outgoing row size is equal or smaller

    int CO_OFF = resting->CO[*(resting->columnSize)];

    int row = *(resting->rowSize);
    int col = *(resting->columnSize) + *(out->columnSize);
    int nonzeros = *(resting->nonzeros) + *(out->nonzeros);

    int counter = 0;

    CSC* target = initCSC(row, col, nonzeros);

    // CO
    for (int i = 0; i < *(resting->columnSize) + 1; i++) {
        target->CO[i] = resting->CO[i];
    }

    for (int i = *(resting->columnSize); i < col + 1; i++) {
        target->CO[i] = CO_OFF + out->CO[counter];
        counter++;
    }

    counter = 0;
    //RI
    for (int i = 0; i < *(resting->nonzeros); i++) {
        target->RI[i] = resting->RI[i];
    }

    for (int i = *(resting->nonzeros); i < nonzeros; i++) {
        target->RI[i] = out->RI[counter];
        counter++;
    }

    counter = 0;
    //DATA
    for (int i = 0; i < *(resting->nonzeros); i++) {
        target->data[i] = resting->data[i];
    }

    for (int i = *(resting->nonzeros); i < nonzeros; i++) {
        target->data[i] = out->data[counter];
        counter++;
    }
    deleteCSC(resting);
    resting = target;
}

void Synapse::connectIn(FLIF* incoming,
    float strength,
    float inhibitory) {

    /*(this->incomingList).push_back(incoming);
    std::vector<std::vector<float>> inWeights;
    initWeights(getN(), incoming->getN(), strength, inhibitory, inWeights);
    addIncomingWeights(this->weights, inWeights);*/

    (this->incomingList).push_back(incoming);
    CSC* inWeights;
    initWeights(incoming->getN(), this->getN(), strength, inhibitory, inWeights);
    addIncomingWeights(this->h_weights, inWeights);
}

void Synapse::connectOut(FLIF* outgoing,
    float strength,
    float inhibitory) {

    /*(this->outgoingList).push_back(outgoing);
    std::vector<std::vector<float>> outWeights;
    initWeights(outgoing->getN(), getN(), strength, inhibitory, outWeights);
    addOutgoingWeights(this->weights, outWeights);*/

    (this->outgoingList).push_back(outgoing);
    CSC* outWeights;
    initWeights(this->getN(), outgoing->getN(), strength, inhibitory, outWeights);
    addOutgoingWeights(this->h_weights, outWeights);
}

void Synapse::connect(Synapse* pre_synaptic, float pre_strength, float pre_inhibitory,
    Synapse* post_synaptic, float post_strength, float post_inhibitory) {
    post_synaptic->connectIn(pre_synaptic, pre_strength, pre_inhibitory);
    pre_synaptic->connectOut(post_synaptic, post_strength, post_inhibitory);
}

void Synapse::connect_GPU(Synapse* pre_synaptic, float pre_strength, float pre_inhibitory,
    Synapse* post_synaptic, float post_strength, float post_inhibitory) {
    post_synaptic->connectIn(pre_synaptic, pre_strength, pre_inhibitory);
    post_synaptic->connectRestore_GPU();

    pre_synaptic->connectOut(post_synaptic, post_strength, post_inhibitory);
    pre_synaptic->connectRestore_GPU();
    
}

void Synapse::connectRestore_GPU() {
    this->freeCSCDevice(this->d_weights);
    this->initCSCDevice(this->d_weights, this->h_weights);
}

void Synapse::POC() {
    Synapse* syn = new Synapse();
    Synapse* syn2 = new Synapse();
    std::cout << "Nonzeros: " <<*(syn->h_weights->nonzeros) << std::endl;
    std::cout << "CO :" << std::endl;
    for (int i = 0; i <= *(syn->h_weights->columnSize); i++) {
        std::cout << syn->h_weights->CO[i] << " " ;
    }
    std::cout << "\nRI :" << std::endl;
    for (int i = 0; i < *(syn->h_weights->nonzeros); i++) {
        std::cout << syn->h_weights->RI[i] << " ";
    }
    std::cout << "\nData :" << std::endl;
    for (int i = 0; i < *(syn->h_weights->nonzeros); i++) {
        std::cout << syn->h_weights->data[i] << " ";
    }
    std::vector<std::vector<float>> target;
    syn->CSCToDense(target, syn->h_weights);
    std::vector<std::vector<float>>::iterator it_w;

    std::cout << std::endl << "DENSE :" << std::endl;
    for (it_w = target.begin(); it_w < target.end(); it_w++) {
        std::cout <<  "|" << syn->vectorToString<float>(*it_w) << "|\n";
    }
    std::cout << "(2,3): " <<syn->getDataCSC(syn->h_weights, 2,3) << std::endl;
    bool* pre;
    pre = new bool[6];
    for (int i = 0; i < 6; i++) {
        pre[i] = true;
    }
    bool* post;
    post = new bool[4];
    for (int i = 0; i < 4; i++) {
        post[i] = true;
    }
    //syn->updateWeights(syn->h_weights, 6, pre, 4, post, 0.2f, 0.0f, 0.0f);

    for (int i = 0; i <= *(syn2->h_weights->columnSize); i++) {
        std::cout << syn2->h_weights->CO[i] << " ";
    }
    std::cout << "\nRI :" << std::endl;
    for (int i = 0; i < *(syn2->h_weights->nonzeros); i++) {
        std::cout << syn2->h_weights->RI[i] << " ";
    }
    std::cout << "\nData :" << std::endl;
    for (int i = 0; i < *(syn2->h_weights->nonzeros); i++) {
        std::cout << syn2->h_weights->data[i] << " ";
    }

    syn->CSCToDense(target, syn2->h_weights);


    std::cout << std::endl << "DENSE :" << std::endl;
    for (it_w = target.begin(); it_w < target.end(); it_w++) {
        std::cout << "|" << syn2->vectorToString<float>(*it_w) << "|\n";
    }
    std::cout << "(2,3): " << syn2->getDataCSC(syn2->h_weights, 2, 3) << std::endl;



    //syn->addOutgoingWeights(syn->h_weights, syn2->h_weights);
    Synapse::connect(syn, 0.4, 0.1, syn2, 0.6, 0.2);
    std::cout << "Nonzeros: " << *(syn->h_weights->nonzeros) << std::endl;
    std::cout << "CO :" << std::endl;
    for (int i = 0; i <= *(syn->h_weights->columnSize); i++) {
        std::cout << syn->h_weights->CO[i] << " ";
    }
    std::cout << "\nRI :" << std::endl;
    for (int i = 0; i < *(syn->h_weights->nonzeros); i++) {
        std::cout << syn->h_weights->RI[i] << " ";
    }
    std::cout << "\nData :" << std::endl;
    for (int i = 0; i < *(syn->h_weights->nonzeros); i++) {
        std::cout << syn->h_weights->data[i] << " ";
    }

    syn->CSCToDense(target, syn->h_weights);


    std::cout << std::endl << "DENSE :" << std::endl;
    for (it_w = target.begin(); it_w < target.end(); it_w++) {
        std::cout << "|" << syn->vectorToString<float>(*it_w) << "|\n";
    }
    std::cout << "(2,3): " << syn->getDataCSC(syn->h_weights, 2, 3) << std::endl;


    syn->deleteCSC(syn->h_weights);



   /* syn->initWeights(8, 8, 0.2f, 0.0f, syn->h_weights);


    std::cout << "Nonzeros: " << syn->h_weights->nonzeros << std::endl;
    std::cout << "CO :" << std::endl;
    for (int i = 0; i <= syn->h_weights->columnSize; i++) {
        std::cout << syn->h_weights->CO[i] << " ";
    }
    std::cout << "\nRI :" << std::endl;
    for (int i = 0; i < syn->h_weights->nonzeros; i++) {
        std::cout << syn->h_weights->RI[i] << " ";
    }
    std::cout << "\nData :" << std::endl;
    for (int i = 0; i < syn->h_weights->nonzeros; i++) {
        std::cout << syn->h_weights->data[i] << " ";
    }

    syn->CSCToDense(target, syn->h_weights);

    std::cout << std::endl << "DENSE :" << std::endl;
    for (it_w = target.begin(); it_w < target.end(); it_w++) {
        std::cout << "|" << syn->vectorToString<float>(*it_w) << "|\n";
    }
    std::cout << std::endl;
    syn->deleteCSC(syn->h_weights);*/

    //std::cout << syn->vectorToString<int>(vec);
}